#include "hip/hip_runtime.h"
#include "environment.h"
#include <iostream>
#include "display.h"
#include <GL/glut.h>
#include <vector>
#include <cmath>
#include <random>
#include <Eigen/Dense>
#include <ctime>
#include <hip/hip_runtime.h>
#include <>
#include <device_types.h>
#include <hip/hip_runtime.h>

using namespace Eigen;

extern const float SCALE_FACTOR = 40;
float scaledWindowWidth = 8;
float scaledWindowHeight = 8;
float windowWidth = scaledWindowWidth * SCALE_FACTOR;
float windowHeight = scaledWindowHeight * SCALE_FACTOR;

//float t = 0; // current time
float dt = 0.001f; // time step
int n_c = 25; // total amt of learner matter
int n_a = 10; // total amt of teacher matter
int totalCount = n_c + n_a;

float pl = 60;
float rotDif = V / pl;
float transDif = pow(RADIUS, 2) * rotDif / 3;
float rotDifCoef = sqrt(2 * rotDif);
float transDifCoef = sqrt(2 * transDif);
float alpha = transDif / rotDif * pow(RADIUS, 2);
float mu = alpha * RADIUS / pl;
float range = 3;
float c = 1;
float s = 1;

std::default_random_engine generator;
std::normal_distribution<double> distribution(0, 1);

float InteractionForce(float);
float Distance(float, float);
float DistancePBC(Matter, Matter, float&, float&);
float RadiansDifference(float, float);
void DrawMatter(Matter, float = 0, float = 0);
Vector2f RandomOrt();
Vector2f RandomPos();

Environment::Environment(float Cnum, float Anum, bool transientEnabled)
{
	srand(time(NULL));
	n_c = Cnum;
	n_a = Anum;
	totalCount = n_c + n_a;

	std::cout << "n_c: " << n_c << std::endl;
	std::cout << "n_a: " << n_a << std::endl;
	std::cout << "all: " << totalCount << std::endl;

	for (int i = 0; i < sqrt(totalCount); i++) {
		for (int j = 0; j < sqrt(totalCount); j++) {
			if (Anum + Cnum == 0)
				break;
			Vector2f ort = RandomOrt();
			float x = i - (windowWidth / 2 / SCALE_FACTOR / 2);
			float y = j - (windowHeight / 2 / SCALE_FACTOR / 2);
			if (Anum > 0) {
				AddMatter(adversarial, x, y, ort(0), ort(1));
				Anum--;
			} else {
				AddMatter(collective, x, y, ort(0), ort(1));
				Cnum--;
			}
		}
	}

	// transient phase
	if (transientEnabled) {
		while (t < 10) {
			for (int i = 0; i < matters.size(); i++) {
				Matter &p = matters[i];
				Movement(p, 0);
			}
			t += dt;
			prevMatters = matters;
		}
		for (int i = 0; i < matters.size(); i++) {
			Matter &p = matters[i];
			p.acmlActiveWork = 0;
		}
		t = 0;
	}
}

std::vector<float> Environment::ReturnState()
{
	std::vector<float> state;
	for (int i = n_a; i < matters.size(); i++) {
		Matter &p = matters[i];

		int inRangeCount = 0;
		float ortx = 0;
		float orty = 0;
		
		for (int j = 0; j < prevMatters.size(); j++)
		{
			Matter &m = prevMatters[j];
			if (m.type == adversarial)
				continue;

			float dx = 0;
			float dy = 0;
			float dis = DistancePBC(p, m, dx, dy);

			if (dis == 0)
				continue;

			if (dis < range) {
				ortx += m.ort[0];
				orty += m.ort[1];
				inRangeCount++;
			}
		}
		float radDiff = 0;
		if (inRangeCount != 0)
			radDiff = RadiansDifference(atan2(orty, ortx), atan2(p.ort[1], p.ort[0]));

		state.push_back(radDiff);
	}
	return state;
}

std::vector<float> Environment::ReturnCState()
{
	std::vector<float> state;
	for (int i = 0; i < n_a; i++) {
		Matter& p = matters[i];

		int inRangeCount = 0;
		float posx = 0;
		float posy = 0;
		
		for (int j = 0; j < prevMatters.size(); j++)
		{
			Matter& m = prevMatters[j];
			if (m.type == adversarial)
				continue;

			float dx = 0;
			float dy = 0;
			float dis = DistancePBC(p, m, dx, dy);

			if (dis == 0)
				continue;

			if (dis < range) {
				posx -= dx;
				posy -= dy;
				inRangeCount++;
			}
		}
		float radDiff = 0;
		if (inRangeCount != 0) {
			posx = (posx) / (float)inRangeCount;
			posy = (posy) / (float)inRangeCount;
			radDiff = RadiansDifference(atan2(posy, posx), atan2(p.ort[1], p.ort[0]));
		}

		state.push_back(radDiff);
	}
	return state;
}

std::vector<float> Environment::ReturnAllState() {
	std::vector<float> jointState = ReturnCState();
	std::vector<float> state_c = ReturnState();
	jointState.insert(jointState.end(), state_c.begin(), state_c.end());
	return jointState;
}

std::vector<float> Environment::Step(std::vector<float> actionList, std::vector<float> &rewardList, bool &terminate)
{
	for (int i = 0; i < matters.size(); i++) {
		Matter &p = matters[i];
		float a = actionList[i];
		Movement(p, a);
	}

	t += dt;
	prevMatters = matters;

	#pragma region Compute active work
	float normActiveWork =  returnCurrentActiveWork();
	float scaledActiveWork = normActiveWork / dt;
	if (normActiveWork > 2 || normActiveWork < -2) terminate = true;


	float activeWork_a = scaledActiveWork * -s;
	std::fill(rewardList.begin(), rewardList.begin() + n_a, activeWork_a);
	float activeWork_c = scaledActiveWork * s;
	std::fill(rewardList.begin() + n_a, rewardList.end(), activeWork_c);
	for (int i = 0; i < n_a; i++) {
		Matter &p = matters[i];
		rewardList[i] += (p.acmlCurrentActiveWork / dt) * -s;
	}
	for (int i = n_a; i < matters.size(); i++) {
		Matter& p = matters[i];
		rewardList[i] += (p.acmlCurrentActiveWork / dt) * s;
	}
	#pragma endregion

	return ReturnAllState();
}

void Environment::Movement(Matter &p, float action) {
	float eta = (float)(distribution(generator));
	float xi_1 = (float)(distribution(generator));
	float xi_2 = (float)(distribution(generator));

	float avgOrt = 0;

	Vector2f r(p.pos[0], p.pos[1]);
	Vector2f rPrev(p.pos[0], p.pos[1]);
	Vector2f tranNoise(xi_1, xi_2);
	Vector2f F(0, 0);

	for (int i = 0; i < prevMatters.size(); i++) {
		Matter &m = prevMatters[i];

		float dx = 0;
		float dy = 0;
		float dis = DistancePBC(p, m, dx, dy);

		if (dis == 0)
			continue;

		float Fr = InteractionForce(dis);
		float forceAngle = atan2(dy, dx);
		F(0) += Fr * cos(forceAngle);
		F(1) += Fr * sin(forceAngle);
	}
	float rad = atan2(p.ort[1], p.ort[0]); // convert ort vector to radians
	float theta = rad + action * dt * c + sqrt(dt) * (rotDifCoef * eta);
	//theta = rad + action * dt * c;

	Vector2f u(cos(theta), sin(theta)); // convert theta to ort vector u

	r = r + dt * (mu * F) + dt * (p.v * u) + sqrt(dt) * (transDifCoef * tranNoise);
	//r = r + dt * (mu * F) + dt * (p.v * u);

	#pragma region Compute active work
	Vector2f r_aw(p.pos[0], p.pos[1]);
	r_aw = (mu * F) + (p.v * u) + (transDifCoef * tranNoise);
	//r_aw = (mu * F) + (p.v * u);
	float aw = dt * (r_aw.dot(u));
	p.acmlActiveWork += aw;
	p.acmlCurrentActiveWork = aw;
	#pragma endregion

	#pragma region PBC Logic
	if (r(0) < -scaledWindowWidth / 2) {
		r(0) += scaledWindowWidth;
		p.posMultiplier[0]--;
	} else if (r(0) >= scaledWindowWidth / 2) {
		r(0) -= scaledWindowWidth;
		p.posMultiplier[0]++;
	}

	if (r(1) < -scaledWindowHeight / 2) {
		r(1) += scaledWindowHeight;
		p.posMultiplier[1]--;
	} else if (r(1) >= scaledWindowHeight / 2) {
		r(1) -= scaledWindowHeight;
		p.posMultiplier[1]++;
	}
	#pragma endregion

	p.pos[0] = r(0);
	p.pos[1] = r(1);
	p.ort[0] = u(0);
	p.ort[1] = u(1);
}

float InteractionForce(float r) {
	if (r < 1.122462f)
		return 48 * pow(1 / r, 13) - 24 * pow((1 / r), 7);
	else
		return 0;
}

float Distance(float dx, float dy)
{
	return sqrt(pow(dx, 2) + pow(dy, 2));
}

float DistancePBC(Matter m1, Matter m2, float &dx, float &dy) {
	dx = m1.pos[0] - m2.pos[0];
	dy = m1.pos[1] - m2.pos[1];

	#pragma region PBC Logic
	if (dx > scaledWindowWidth / 2)
		dx -= scaledWindowWidth;
	else if (dx <= -scaledWindowWidth / 2)
		dx += scaledWindowWidth;

	if (dy > scaledWindowHeight / 2)
		dy -= scaledWindowHeight;
	else if (dy <= -scaledWindowHeight / 2)
		dy += scaledWindowHeight;
	#pragma endregion

	return Distance(dx, dy);
}

float RadiansDifference(float radA, float radB) {
	radA += M_PI;
	radB += M_PI;
	float d = fmodf(abs(radA - radB), (float)M_PI*2);
	float r = d > M_PI ? M_PI*2 - d : d;
	if ((radA - radB >= 0 && radA - radB <= M_PI) || (radA - radB <= -M_PI && radA - radB >= -M_PI*2)) 
		return r;
	return -r;
}

float Environment::returnAllActiveWork() {
	float totalActiveWork = 0;
	for (int i = 0; i < matters.size(); i++) {
		Matter &p = matters[i];
		totalActiveWork += p.acmlActiveWork;
	}
	return (1 / ((float)(n_c+n_a)*t)) * totalActiveWork;
}

float Environment::returnActiveWork_c() {
	float totalActiveWork = 0;
	for (int i = n_a; i < matters.size(); i++) {
		Matter& p = matters[i];
		totalActiveWork += p.acmlActiveWork;
	}
	return (1 / ((float)(n_c) * t)) * totalActiveWork;
}

float Environment::returnActiveWork_a() {
	float totalActiveWork = 0;
	for (int i = 0; i < n_a; i++) {
		Matter& p = matters[i];
		totalActiveWork += p.acmlActiveWork;
	}
	return (1 / ((float)(n_a)*t)) * totalActiveWork;
}

float Environment::returnCurrentActiveWork() {
	float totalActiveWork = 0;
	for (int i = 0; i < matters.size(); i++) {
		Matter &p = matters[i];
		totalActiveWork += p.acmlCurrentActiveWork;
	}
	return (1 / (float)(n_c + n_a)) * totalActiveWork;
}

void Environment::Display()
{
	glClear(GL_COLOR_BUFFER_BIT);

	//draw matters
	for (int i = 0; i < matters.size(); i++) {
		Matter &p = matters[i];
		DrawMatter(p);

		#pragma region PBC Logic
		if (p.pos[0] - p.r / 2 < -scaledWindowWidth / 2) {
			DrawMatter(p, +scaledWindowWidth, 0);
		}
		else if (p.pos[0] + p.r / 2 >= scaledWindowWidth / 2) {
			DrawMatter(p, -scaledWindowWidth, 0);
		}

		if (p.pos[1] - p.r / 2 < -scaledWindowHeight / 2) {
			DrawMatter(p, 0, scaledWindowHeight);
		}
		else if (p.pos[1] + p.r / 2 >= scaledWindowHeight / 2) {
			DrawMatter(p, 0, -scaledWindowHeight);
		}
		#pragma endregion
	}

	glFlush();
	glutSwapBuffers();
}

void DrawMatter(Matter p, float transformX, float transformY) {
	glPushMatrix();
	glScalef(SCALE_FACTOR, SCALE_FACTOR, 1.0);
	glTranslatef(p.pos[0] + transformX, p.pos[1] + transformY, 0.0f);

	glColor3f(p.ort[0], p.ort[1], 1);

	glBegin(GL_POLYGON);
	for (float a = 0; a < 2 * M_PI; a += 0.2)
		glVertex2f(p.r / 2 * cos(a), p.r / 2 * sin(a));
	glEnd();

	float degree = -(atan2(p.ort[0], p.ort[1]) * 180 / M_PI);
	glRotatef(degree, 0.0f, 0.0f, 1.0f);

	glColor3f(0, 0, 0);
	glBegin(GL_TRIANGLES);
	glVertex2f(0, 0.45);
	glVertex2f(-0.25, -0.25);
	glVertex2f(0.25, -0.25);
	glEnd();

	glPopMatrix();
}

Vector2f RandomPos() {
	int randSignX = (rand() > RAND_MAX / 2) ? -1 : 1;
	float x = randSignX * (rand() / (RAND_MAX / (floor(windowWidth / SCALE_FACTOR) / 2)));
	int randSignY = (rand() > RAND_MAX / 2) ? -1 : 1;
	float y = randSignY * (rand() / (RAND_MAX / (floor(windowHeight / SCALE_FACTOR) / 2)));

	Vector2f pos(x, y);
	return pos;
}

Vector2f RandomOrt() {
	int randTotal = (rand() > RAND_MAX / 2) ? -1 : 1;
	float x = 2 * (float)rand() / (float)RAND_MAX - 1;

	float y = randTotal - x;
	Vector2f ort(x, y);
	return ort;
}

void Environment::AddMatter(MatterType mt)
{
	Matter p;
	Vector2f pos = RandomPos();
	p.pos[0] = pos(0);
	p.pos[1] = pos(1);
	p.v = V;
	p.r = RADIUS;
	Vector2f ort = RandomOrt();
	p.ort[0] = ort(0);
	p.ort[1] = ort(1);
	p.type = mt;
	matters.push_back(p);
	prevMatters.push_back(p);
}

void Environment::AddMatter(MatterType mt, float x, float y, float ortx, float orty)
{
	Matter p;
	p.pos[0] = x;
	p.pos[1] = y;
	p.v = V;
	p.r = RADIUS;
	p.ort[0] = ortx;
	p.ort[1] = orty;
	p.type = mt;
	matters.push_back(p);
	prevMatters.push_back(p);
}

void Environment::RemoveMatters()
{
	matters.clear();
	prevMatters.clear();
}

Environment::~Environment()
{
	RemoveMatters();
}