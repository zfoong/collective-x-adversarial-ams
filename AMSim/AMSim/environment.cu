#include "hip/hip_runtime.h"
#include "environment.cuh"
#include <iostream>
#include "display.cuh"
#include <GL/glut.h>
#include <vector>
#include <cmath>
#include <random>
#include <Eigen/Dense>
#include <ctime>
#include <hip/hip_runtime.h>
#include <>
#include <device_types.h>
#include <hip/device_functions.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

using namespace Eigen;

extern const float SCALE_FACTOR = 40;
float scaledWindowWidth = 12;
float scaledWindowHeight = 12;
float windowWidth = scaledWindowWidth * SCALE_FACTOR;
float windowHeight = scaledWindowHeight * SCALE_FACTOR;

float t = 0; // current time
float dt = 0.01f; // time step
int n = 1; // total amt of learner matter
int n_t = 32; // total amt of teacher matter
int totalCount = n + n_t;

float pl = 60;
float rotDif = V / pl;
float transDif = pow(RADIUS, 2) * rotDif / 3;
float rotDifCoef = sqrt(2 * rotDif);
float transDifCoef = sqrt(2 * transDif);
float alpha = transDif / rotDif * pow(RADIUS, 2);
float mu = alpha * RADIUS / pl;
float range = 3;
float c = 1;
float s = 3;

float env_pi = 3.14159265358979323846;

std::default_random_engine generator;
std::normal_distribution<double> distribution(0, 1);

__device__ float InteractionForce(float);
float Distance(float, float);
float DistancePBC(Matter, Matter, float&, float&);
float RadiansDifference(float, float);
__global__ void global_Movement(Matter*, Matter*, float*, int, float);
void DrawMatter(Matter, float = 0, float = 0);
Vector2f RandomOrt();
Vector2f RandomPos();

__device__ float d_mu;
__device__ float d_rotDifCoef;
__device__ float d_transDifCoef;
__device__ float d_sWidth;
__device__ float d_sHeight;
__device__ float d_range;

Environment::Environment(float Lnum, float Tnum, bool transientEnabled)
{
	hipMemcpyToSymbol(HIP_SYMBOL(&d_mu), &mu, sizeof(float*), size_t(0), hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(&d_rotDifCoef), &rotDifCoef, sizeof(float*), size_t(0), hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(&d_transDifCoef), &transDifCoef, sizeof(float*), size_t(0), hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(&d_sWidth), &scaledWindowWidth, sizeof(float*), size_t(0), hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(&d_sHeight), &scaledWindowHeight, sizeof(float*), size_t(0), hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(&d_range), &range, sizeof(float*), size_t(0), hipMemcpyHostToDevice);

	srand(time(NULL));
	n = Lnum;
	n_t = Tnum;
	totalCount = n + n_t;
	for (int i = 0; i < sqrt(totalCount); i++) {
		for (int j = 0; j < sqrt(totalCount); j++) {
			if (Tnum + Lnum == 0)
				break;
			Vector2f ort = RandomOrt();
			float x = i - (windowWidth / 2 / SCALE_FACTOR / 2);
			float y = j - (windowHeight / 2 / SCALE_FACTOR / 2);
			if (Tnum > 0) {
				AddMatter(teacher, x, y, ort(0), ort(1));
				Tnum--;
			} else {
				AddMatter(learner, x, y, ort(0), ort(1));
				Lnum--;
			}
		}
	}

	// transient phase
	/*if (transientEnabled) {
		while (t < 10) {
			for (int i = 0; i < matters.size(); i++) {
				Matter &p = matters[i];
				Movement(p, 0);
			}
			t += dt;
			prevMatters = matters;
		}
		for (int i = 0; i < matters.size(); i++) {
			Matter &p = matters[i];
			p.acmlActiveWork = 0;
		}
		t = 0;
	}*/
}

std::vector<float> Environment::ReturnState()
{
	std::vector<float> state;
	for (int i = n_t; i < matters.size(); i++) {
		Matter &p = matters[i];

		int inRangeCount = 0;
		float totalRad = 0;
		float ortx = 0;
		float orty = 0;
		for (int j = 0; j < prevMatters.size(); j++)
		{
			Matter &m = prevMatters[j];
			float dx = 0;
			float dy = 0;
			float dis = DistancePBC(p, m, dx, dy);

			if (dis == 0)
				continue;

			if (dis < range) {
				ortx += m.ort[0];
				orty += m.ort[1];
				inRangeCount++;
			}
		}
		float radDiff = 0;
		if (inRangeCount != 0)
			radDiff = RadiansDifference(atan2(orty, ortx), atan2(p.ort[1], p.ort[0]));

		state.push_back(radDiff);
	}
	return state;
}

std::vector<float> Environment::Step(std::vector<float> actionList, std::vector<float> &rewardList, bool &terminate)
{
	Movement(actionList);

	t += dt;
	prevMatters = matters;

	//for (int i = n_t; i < matters.size(); i++) {
	//	Matter &p = matters[i];

	//	#pragma region Compute reward by neighbour lost
	//	int inRangeCount = 0;
	//	for (int j = 0; j < prevMatters.size(); j++) {
	//		Matter &m = prevMatters[j];
	//		float dx = 0;
	//		float dy = 0;
	//		float dis = DistancePBC(p, m, dx, dy);

	//		if (dis == 0)
	//			continue;

	//		if (dis < range)
	//			inRangeCount++;
	//	}

	//	if (inRangeCount < p.neighbourCount)
	//		rewardList[i-n_t] = (inRangeCount - p.neighbourCount)*c;
	//	else
	//		rewardList[i-n_t] = 0;
	//	p.neighbourCount = inRangeCount;
	//	#pragma endregion

	//}

	#pragma region Compute active work
	float normActiveWork =  returnCurrentActiveWork();
	float scaledActiveWork = powf((normActiveWork / dt)*10, 2);
	if (normActiveWork > 2 || normActiveWork < -2) terminate = true;
	//float G = 1 / dt * std::log(std::exp(-s*dt*totalCount*scaledActiveWork));
	//float activeWork_s = -(G / totalCount);

	std::fill(rewardList.begin(), rewardList.end(), scaledActiveWork);
	//for (int i = n_t; i < matters.size(); i++) {
	//	Matter &p = matters[i];
	//	rewardList[i - n_t] += powf((p.acmlCurrentActiveWork / dt) * 10, 2);;
	//}
	#pragma endregion

	return ReturnState();
}


void Environment::Movement(std::vector<float> actionList) {

	int count = matters.size();
	float* h_action;
	h_action = (float*)malloc(count * sizeof(float));
	for (int i = 0; i < matters.size(); i++) {
		h_action[i] = matters[i].type == learner ? actionList[i - n_t] : 0;
	}

	Matter* d_matters;
	Matter* h_matters;
	h_matters = (Matter*)malloc(count * sizeof(Matter));
	h_matters = &matters[0];
	Matter* d_prevMatters;
	Matter* h_prevMatters;
	h_prevMatters = (Matter*)malloc(count * sizeof(Matter));
	h_prevMatters = &prevMatters[0];
	float* d_action;

	std::cout << count * sizeof(Matter) << std::endl;

	if(hipMalloc((void**)&d_action, count * sizeof(float)) != hipSuccess) std::cout <<  hipGetErrorString(hipGetLastError()) << std::endl;
	if(hipMalloc((void **)&d_matters, count * sizeof(Matter)) != hipSuccess) std::cout << "failed 1!" << std::endl;
	if(hipMalloc((void**)&d_prevMatters, count * sizeof(Matter)) != hipSuccess) std::cout << "failed 2!" << std::endl;

	std::cout << count * sizeof(Matter) << std::endl;

	if(hipMemcpy(d_matters, h_matters, count * sizeof(Matter), hipMemcpyHostToDevice) != hipSuccess) std::cout << "mem1 copy failed!" << std::endl;
	if(hipMemcpy(d_prevMatters, h_prevMatters, count * sizeof(Matter), hipMemcpyHostToDevice) != hipSuccess) std::cout << "mem2 copy failed!" << std::endl;
	if(hipMemcpy(d_action, h_action, count * sizeof(Matter), hipMemcpyHostToDevice) != hipSuccess) std::cout << "mem3 copy failed!" << std::endl;

	global_Movement<<<(count + 255) / 256, 256 >>>(d_matters, d_prevMatters, d_action, count, 0.01);

	if(hipMemcpy(h_matters, d_matters, count * sizeof(Matter), hipMemcpyDeviceToHost) != hipSuccess) std::cout << "mem copy out failed!" << std::endl;
	matters.assign(h_matters, h_matters + count);


	hipFree(d_matters);
	hipFree(d_prevMatters);
	hipFree(d_action);
	//free(h_matters);
}

__global__ void global_Movement(Matter* d_matters, Matter* d_prevMatters, float* d_action, int count, float d_dt) {

	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < count) {
		Matter p = d_matters[i];
		float action = d_action[i];

		hiprandState state;
		hiprand_init(0, i, 0, &state);
		float eta = hiprand_normal(&state);
		float xi_1 = hiprand_normal(&state);
		float xi_2 = hiprand_normal(&state);

		float avgOrt = 0;
		float ortx = 0;
		float orty = 0;

		float r_0 = p.pos[0];
		float r_1 = p.pos[1];
		float F_0 = 0;
		float F_1 = 1;

		for (int j = 0; j < count; j++) {
			Matter m = d_prevMatters[j];

			float dx = p.pos[0] - m.pos[0];
			float dy = p.pos[1] - m.pos[1];

			#pragma region PBC Logic
			if (dx > d_sWidth / 2)
				dx -= d_sWidth;
			else if (dx <= -d_sWidth / 2)
				dx += d_sWidth;

			if (dy > d_sHeight / 2)
				dy -= d_sHeight;
			else if (dy <= -d_sHeight / 2)
				dy += d_sHeight;
			#pragma endregion

			float dis = sqrt(pow(dx, 2) + pow(dy, 2));

			if (dis == 0)
				continue;

			if (dis < d_range && m.type != learner) {
				ortx += m.ort[0];
				orty += m.ort[1];
			}

			float Fr = InteractionForce(dis);
			float forceAngle = atan2(dy, dx);
			F_0 += Fr * cos(forceAngle);
			F_1 += Fr * sin(forceAngle);
		}
		float rad = atan2(p.ort[1], p.ort[0]); // convert ort vector to radians
		float theta = 0;
		if (p.type == teacher) {
			float radDiff = 0;

			float radA = atan2(orty, ortx);
			float radB = rad;

			float pi = 3.14159265358979323846;

			radA += pi;
			radB += pi;
			float d = fmodf(abs(radA - radB), (float)pi * 2);
			float r = d > pi ? pi * 2 - d : d;
			if ((radA - radB >= 0 && radA - radB <= pi) || (radA - radB <= -pi && radA - radB >= -pi * 2))
				radDiff = r;
			radDiff = -r;

			if (ortx == 0 && orty == 0)
				radDiff = 0;
			theta = rad + radDiff * d_dt + sqrt(d_dt) * (d_rotDifCoef * eta);
		}
		else {
			theta = rad + action * d_dt + sqrt(d_dt) * (d_rotDifCoef * eta);
		}

		float u_0 = cos(theta);
		float u_1 = sin(theta);

		r_0 = r_0 + d_dt * (d_mu * F_0) + d_dt * (p.v * u_0) + sqrt(d_dt) * (d_transDifCoef * xi_1);
		r_1 = r_1 + d_dt * (d_mu * F_1) + d_dt * (p.v * u_1) + sqrt(d_dt) * (d_transDifCoef * xi_2);

		#pragma region Compute active work
		float r_aw_0 = p.pos[0];
		float r_aw_1 = p.pos[1];
		r_aw_0 = (d_mu * F_0) + (p.v * u_0) + (d_transDifCoef * xi_1);
		r_aw_1 = (d_mu * F_1) + (p.v * u_1) + (d_transDifCoef * xi_2);
		float aw = d_dt * (r_aw_0 * u_0 + r_aw_1 * u_1);
		p.acmlActiveWork += aw;
		p.acmlCurrentActiveWork = aw;
		#pragma endregion

		#pragma region PBC Logic
		if (r_0 < -d_sWidth / 2) {
			r_0 += d_sWidth;
			p.posMultiplier[0]--;
		}
		else if (r_0 >= d_sWidth / 2) {
			r_0 -= d_sWidth;
			p.posMultiplier[0]++;
		}

		if (r_1 < -d_sHeight / 2) {
			r_1 += d_sHeight;
			p.posMultiplier[1]--;
		}
		else if (r_1 >= d_sHeight / 2) {
			r_1 -= d_sHeight;
			p.posMultiplier[1]++;
		}
		#pragma endregion

		p.pos[0] = r_0;
		p.pos[1] = r_1;
		p.ort[0] = u_0;
		p.ort[1] = u_1;
	}
}

__device__ float InteractionForce(float r) {
	if (r < 1.122462f)
		return 48 * pow(1 / r, 13) - 24 * pow((1 / r), 7);
	else
		return 0;
}

float Distance(float dx, float dy)
{
	return sqrt(pow(dx, 2) + pow(dy, 2));
}

float DistancePBC(Matter m1, Matter m2, float &dx, float &dy) {
	dx = m1.pos[0] - m2.pos[0];
	dy = m1.pos[1] - m2.pos[1];

	#pragma region PBC Logic
	if (dx > scaledWindowWidth / 2)
		dx -= scaledWindowWidth;
	else if (dx <= -scaledWindowWidth / 2)
		dx += scaledWindowWidth;

	if (dy > scaledWindowHeight / 2)
		dy -= scaledWindowHeight;
	else if (dy <= -scaledWindowHeight / 2)
		dy += scaledWindowHeight;
	#pragma endregion

	return Distance(dx, dy);
}

float RadiansDifference(float radA, float radB) {
	radA += env_pi;
	radB += env_pi;
	float d = fmodf(abs(radA - radB), (float)env_pi*2);
	float r = d > env_pi ? env_pi*2 - d : d;
	if ((radA - radB >= 0 && radA - radB <= env_pi) || (radA - radB <= -env_pi && radA - radB >= -env_pi*2)) 
		return r;
	return -r;
}

float Environment::returnActiveWork() {
	float totalActiveWork = 0;
	for (int i = 0; i < matters.size(); i++) {
		Matter &p = matters[i];
		totalActiveWork += p.acmlActiveWork;
	}
	return (1 / ((float)(n+n_t)*t)) * totalActiveWork;
}

float Environment::returnCurrentActiveWork() {
	float totalActiveWork = 0;
	for (int i = 0; i < matters.size(); i++) {
		Matter &p = matters[i];
		totalActiveWork += p.acmlCurrentActiveWork;
	}
	return (1 / (float)(n + n_t)) * totalActiveWork;
}

void Environment::Display()
{
	glClear(GL_COLOR_BUFFER_BIT);

	//draw matters
	for (int i = 0; i < matters.size(); i++) {
		Matter &p = matters[i];
		DrawMatter(p);

		#pragma region PBC Logic
		if (p.pos[0] - p.r / 2 < -scaledWindowWidth / 2) {
			DrawMatter(p, +scaledWindowWidth, 0);
		}
		else if (p.pos[0] + p.r / 2 >= scaledWindowWidth / 2) {
			DrawMatter(p, -scaledWindowWidth, 0);
		}

		if (p.pos[1] - p.r / 2 < -scaledWindowHeight / 2) {
			DrawMatter(p, 0, scaledWindowHeight);
		}
		else if (p.pos[1] + p.r / 2 >= scaledWindowHeight / 2) {
			DrawMatter(p, 0, -scaledWindowHeight);
		}
		#pragma endregion
	}

	glFlush();
	glutSwapBuffers();
}

void DrawMatter(Matter p, float transformX, float transformY) {
	glPushMatrix();
	glScalef(SCALE_FACTOR, SCALE_FACTOR, 1.0);
	glTranslatef(p.pos[0] + transformX, p.pos[1] + transformY, 0.0f);

	glColor3f(p.ort[0], p.ort[1], 1);

	glBegin(GL_POLYGON);
	for (float a = 0; a < 2 * env_pi; a += 0.2)
		glVertex2f(p.r / 2 * cos(a), p.r / 2 * sin(a));
	glEnd();

	float degree = -(atan2(p.ort[0], p.ort[1]) * 180 / env_pi);
	glRotatef(degree, 0.0f, 0.0f, 1.0f);

	glColor3f(0, 0, 0);
	glBegin(GL_TRIANGLES);
	glVertex2f(0, 0.45);
	glVertex2f(-0.25, -0.25);
	glVertex2f(0.25, -0.25);
	glEnd();

	glPopMatrix();
}

Vector2f RandomPos() {
	int randSignX = (rand() > RAND_MAX / 2) ? -1 : 1;
	float x = randSignX * (rand() / (RAND_MAX / (floor(windowWidth / SCALE_FACTOR) / 2)));
	int randSignY = (rand() > RAND_MAX / 2) ? -1 : 1;
	float y = randSignY * (rand() / (RAND_MAX / (floor(windowHeight / SCALE_FACTOR) / 2)));

	Vector2f pos(x, y);
	return pos;
}

Vector2f RandomOrt() {
	int randTotal = (rand() > RAND_MAX / 2) ? -1 : 1;
	float x = 2 * (float)rand() / (float)RAND_MAX - 1;

	float y = randTotal - x;
	Vector2f ort(x, y);
	return ort;
}

void Environment::AddMatter(MatterType mt)
{
	Matter p;
	Vector2f pos = RandomPos();
	p.pos[0] = pos(0);
	p.pos[1] = pos(1);
	p.v = V;
	p.r = RADIUS;
	Vector2f ort = RandomOrt();
	p.ort[0] = ort(0);
	p.ort[1] = ort(1);
	p.type = mt;
	matters.push_back(p);
	prevMatters.push_back(p);
}

void Environment::AddMatter(MatterType mt, float x, float y, float ortx, float orty)
{
	Matter p;
	p.pos[0] = x;
	p.pos[1] = y;
	p.v = V;
	p.r = RADIUS;
	p.ort[0] = ortx;
	p.ort[1] = orty;
	p.type = mt;
	matters.push_back(p);
	prevMatters.push_back(p);
}

void Environment::RemoveMatters()
{
	matters.clear();
	prevMatters.clear();
}

Environment::~Environment()
{
	RemoveMatters();
}