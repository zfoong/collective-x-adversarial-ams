#include "hip/hip_runtime.h"
#include "environment.h"
#include <iostream>
#include "display.h"
#include <GL/glut.h>
#include <vector>
#include <cmath>
#include <random>
#include <Eigen/Dense>
#include <ctime>
#include <hip/hip_runtime.h>
#include <>
#include <device_types.h>
#include <hip/hip_runtime.h>

using namespace Eigen;

extern const float SCALE_FACTOR = 40;
float scaledWindowWidth = 12;
float scaledWindowHeight = 12;
float windowWidth = scaledWindowWidth * SCALE_FACTOR;
float windowHeight = scaledWindowHeight * SCALE_FACTOR;

//float t = 0; // current time
float dt = 0.01f; // time step
int n = 1; // total amt of learner matter
int n_t = 32; // total amt of teacher matter
int totalCount = n + n_t;

float pl = 60;
float rotDif = V / pl;
float transDif = pow(RADIUS, 2) * rotDif / 3;
float rotDifCoef = sqrt(2 * rotDif);
float transDifCoef = sqrt(2 * transDif);
float alpha = transDif / rotDif * pow(RADIUS, 2);
float mu = alpha * RADIUS / pl;
float range = 3;
float c = 1;
float s = 1;

std::default_random_engine generator;
std::normal_distribution<double> distribution(0, 1);

float InteractionForce(float);
float Distance(float, float);
float DistancePBC(Matter, Matter, float&, float&);
float RadiansDifference(float, float);
void DrawMatter(Matter, float = 0, float = 0);
Vector2f RandomOrt();
Vector2f RandomPos();

Environment::Environment(float Lnum, float Tnum, bool transientEnabled)
{
	srand(time(NULL));
	n = Lnum;
	n_t = Tnum;
	totalCount = n + n_t;
	for (int i = 0; i < sqrt(totalCount); i++) {
		for (int j = 0; j < sqrt(totalCount); j++) {
			if (Tnum + Lnum == 0)
				break;
			Vector2f ort = RandomOrt();
			float x = i - (windowWidth / 2 / SCALE_FACTOR / 2);
			float y = j - (windowHeight / 2 / SCALE_FACTOR / 2);
			if (Tnum > 0) {
				AddMatter(teacher, x, y, ort(0), ort(1));
				Tnum--;
			} else {
				AddMatter(learner, x, y, ort(0), ort(1));
				Lnum--;
			}
		}
	}

	// transient phase
	if (transientEnabled) {
		while (t < 10) {
			for (int i = 0; i < matters.size(); i++) {
				Matter &p = matters[i];
				Movement(p, 0);
			}
			t += dt;
			prevMatters = matters;
		}
		for (int i = 0; i < matters.size(); i++) {
			Matter &p = matters[i];
			p.acmlActiveWork = 0;
		}
		t = 0;
	}
}

std::vector<float> Environment::ReturnState()
{
	std::vector<float> state;
	for (int i = n_t; i < matters.size(); i++) {
		Matter &p = matters[i];

		int inRangeCount = 0;
		float totalRad = 0;
		float ortx = 0;
		float orty = 0;
		for (int j = 0; j < prevMatters.size(); j++)
		{
			Matter &m = prevMatters[j];
			float dx = 0;
			float dy = 0;
			float dis = DistancePBC(p, m, dx, dy);

			if (dis == 0)
				continue;

			if (dis < range) {
				ortx += m.ort[0];
				orty += m.ort[1];
				inRangeCount++;
			}
		}
		float radDiff = 0;
		if (inRangeCount != 0)
			radDiff = RadiansDifference(atan2(orty, ortx), atan2(p.ort[1], p.ort[0]));

		state.push_back(radDiff);
	}
	return state;
}

std::vector<float> Environment::Step(std::vector<float> actionList, std::vector<float> &rewardList, bool &terminate)
{
	for (int i = 0; i < matters.size(); i++) {
		Matter &p = matters[i];
		float a = 0;
		if(p.type == learner) a = actionList[i-n_t];
		Movement(p, a);
	}

	t += dt;
	prevMatters = matters;

	//for (int i = n_t; i < matters.size(); i++) {
	//	Matter &p = matters[i];

	//	#pragma region Compute reward by neighbour lost
	//	int inRangeCount = 0;
	//	for (int j = 0; j < prevMatters.size(); j++) {
	//		Matter &m = prevMatters[j];
	//		float dx = 0;
	//		float dy = 0;
	//		float dis = DistancePBC(p, m, dx, dy);

	//		if (dis == 0)
	//			continue;

	//		if (dis < range)
	//			inRangeCount++;
	//	}

	//	if (inRangeCount < p.neighbourCount)
	//		rewardList[i-n_t] = (inRangeCount - p.neighbourCount)*c;
	//	else
	//		rewardList[i-n_t] = 0;
	//	p.neighbourCount = inRangeCount;
	//	#pragma endregion

	//}

	#pragma region Compute active work
	float normActiveWork =  returnCurrentActiveWork();
	float scaledActiveWork = normActiveWork / dt;
	if (normActiveWork > 2 || normActiveWork < -2) terminate = true;
	float activeWork_s = scaledActiveWork * -s;

	std::fill(rewardList.begin(), rewardList.end(), activeWork_s);
	for (int i = n_t; i < matters.size(); i++) {
		Matter &p = matters[i];
		rewardList[i - n_t] += (p.acmlCurrentActiveWork / dt) * -s;
	}
	#pragma endregion

	return ReturnState();
}

void Environment::Movement(Matter &p, float action) {
	float eta = (float)(distribution(generator));
	float xi_1 = (float)(distribution(generator));
	float xi_2 = (float)(distribution(generator));

	float avgOrt = 0;
	float ortx = 0;
	float orty = 0;

	Vector2f r(p.pos[0], p.pos[1]);
	Vector2f rPrev(p.pos[0], p.pos[1]);
	Vector2f tranNoise(xi_1, xi_2);
	Vector2f F(0, 0);

	for (int i = 0; i < prevMatters.size(); i++) {
		Matter &m = prevMatters[i];

		float dx = 0;
		float dy = 0;
		float dis = DistancePBC(p, m, dx, dy);

		if (dis == 0)
			continue;

		if (dis < range && m.type != learner) {
			ortx += m.ort[0];
			orty += m.ort[1];
		}

		float Fr = InteractionForce(dis);
		float forceAngle = atan2(dy, dx);
		F(0) += Fr * cos(forceAngle);
		F(1) += Fr * sin(forceAngle);
	}
	float rad = atan2(p.ort[1], p.ort[0]); // convert ort vector to radians
	float theta = 0;
	if (p.type == teacher) {
		float radDiff = RadiansDifference(atan2(orty, ortx), rad);
		if (ortx == 0 && orty == 0)
			radDiff = 0;
		theta = rad + radDiff * dt + sqrt(dt) * (rotDifCoef * eta);
		//theta = rad + radDiff * dt;
	}
	else {
		theta = rad + action * dt * c + sqrt(dt) * (rotDifCoef * eta);
		//theta = rad + action * dt * c;
	}

	Vector2f u(cos(theta), sin(theta)); // convert theta to ort vector u

	r = r + dt * (mu * F) + dt * (p.v * u) + sqrt(dt) * (transDifCoef * tranNoise);
	//r = r + dt * (mu * F) + dt * (p.v * u);

	#pragma region Compute active work
	Vector2f r_aw(p.pos[0], p.pos[1]);
	r_aw = (mu * F) + (p.v * u) + (transDifCoef * tranNoise);
	//r_aw = (mu * F) + (p.v * u);
	float aw = dt * (r_aw.dot(u));
	p.acmlActiveWork += aw;
	p.acmlCurrentActiveWork = aw;
	#pragma endregion

	#pragma region PBC Logic
	if (r(0) < -scaledWindowWidth / 2) {
		r(0) += scaledWindowWidth;
		p.posMultiplier[0]--;
	} else if (r(0) >= scaledWindowWidth / 2) {
		r(0) -= scaledWindowWidth;
		p.posMultiplier[0]++;
	}

	if (r(1) < -scaledWindowHeight / 2) {
		r(1) += scaledWindowHeight;
		p.posMultiplier[1]--;
	} else if (r(1) >= scaledWindowHeight / 2) {
		r(1) -= scaledWindowHeight;
		p.posMultiplier[1]++;
	}
	#pragma endregion

	p.pos[0] = r(0);
	p.pos[1] = r(1);
	p.ort[0] = u(0);
	p.ort[1] = u(1);
}

float InteractionForce(float r) {
	if (r < 1.122462f)
		return 48 * pow(1 / r, 13) - 24 * pow((1 / r), 7);
	else
		return 0;
}

float Distance(float dx, float dy)
{
	return sqrt(pow(dx, 2) + pow(dy, 2));
}

float DistancePBC(Matter m1, Matter m2, float &dx, float &dy) {
	dx = m1.pos[0] - m2.pos[0];
	dy = m1.pos[1] - m2.pos[1];

	#pragma region PBC Logic
	if (dx > scaledWindowWidth / 2)
		dx -= scaledWindowWidth;
	else if (dx <= -scaledWindowWidth / 2)
		dx += scaledWindowWidth;

	if (dy > scaledWindowHeight / 2)
		dy -= scaledWindowHeight;
	else if (dy <= -scaledWindowHeight / 2)
		dy += scaledWindowHeight;
	#pragma endregion

	return Distance(dx, dy);
}

float RadiansDifference(float radA, float radB) {
	radA += M_PI;
	radB += M_PI;
	float d = fmodf(abs(radA - radB), (float)M_PI*2);
	float r = d > M_PI ? M_PI*2 - d : d;
	if ((radA - radB >= 0 && radA - radB <= M_PI) || (radA - radB <= -M_PI && radA - radB >= -M_PI*2)) 
		return r;
	return -r;
}

float Environment::returnActiveWork() {
	float totalActiveWork = 0;
	for (int i = 0; i < matters.size(); i++) {
		Matter &p = matters[i];
		totalActiveWork += p.acmlActiveWork;
	}
	return (1 / ((float)(n+n_t)*t)) * totalActiveWork;
}

float Environment::returnCurrentActiveWork() {
	float totalActiveWork = 0;
	for (int i = 0; i < matters.size(); i++) {
		Matter &p = matters[i];
		totalActiveWork += p.acmlCurrentActiveWork;
	}
	return (1 / (float)(n + n_t)) * totalActiveWork;
}

void Environment::Display()
{
	glClear(GL_COLOR_BUFFER_BIT);

	//draw matters
	for (int i = 0; i < matters.size(); i++) {
		Matter &p = matters[i];
		DrawMatter(p);

		#pragma region PBC Logic
		if (p.pos[0] - p.r / 2 < -scaledWindowWidth / 2) {
			DrawMatter(p, +scaledWindowWidth, 0);
		}
		else if (p.pos[0] + p.r / 2 >= scaledWindowWidth / 2) {
			DrawMatter(p, -scaledWindowWidth, 0);
		}

		if (p.pos[1] - p.r / 2 < -scaledWindowHeight / 2) {
			DrawMatter(p, 0, scaledWindowHeight);
		}
		else if (p.pos[1] + p.r / 2 >= scaledWindowHeight / 2) {
			DrawMatter(p, 0, -scaledWindowHeight);
		}
		#pragma endregion
	}

	glFlush();
	glutSwapBuffers();
}

void DrawMatter(Matter p, float transformX, float transformY) {
	glPushMatrix();
	glScalef(SCALE_FACTOR, SCALE_FACTOR, 1.0);
	glTranslatef(p.pos[0] + transformX, p.pos[1] + transformY, 0.0f);

	glColor3f(p.ort[0], p.ort[1], 1);

	glBegin(GL_POLYGON);
	for (float a = 0; a < 2 * M_PI; a += 0.2)
		glVertex2f(p.r / 2 * cos(a), p.r / 2 * sin(a));
	glEnd();

	float degree = -(atan2(p.ort[0], p.ort[1]) * 180 / M_PI);
	glRotatef(degree, 0.0f, 0.0f, 1.0f);

	glColor3f(0, 0, 0);
	glBegin(GL_TRIANGLES);
	glVertex2f(0, 0.45);
	glVertex2f(-0.25, -0.25);
	glVertex2f(0.25, -0.25);
	glEnd();

	glPopMatrix();
}

Vector2f RandomPos() {
	int randSignX = (rand() > RAND_MAX / 2) ? -1 : 1;
	float x = randSignX * (rand() / (RAND_MAX / (floor(windowWidth / SCALE_FACTOR) / 2)));
	int randSignY = (rand() > RAND_MAX / 2) ? -1 : 1;
	float y = randSignY * (rand() / (RAND_MAX / (floor(windowHeight / SCALE_FACTOR) / 2)));

	Vector2f pos(x, y);
	return pos;
}

Vector2f RandomOrt() {
	int randTotal = (rand() > RAND_MAX / 2) ? -1 : 1;
	float x = 2 * (float)rand() / (float)RAND_MAX - 1;

	float y = randTotal - x;
	Vector2f ort(x, y);
	return ort;
}

void Environment::AddMatter(MatterType mt)
{
	Matter p;
	Vector2f pos = RandomPos();
	p.pos[0] = pos(0);
	p.pos[1] = pos(1);
	p.v = V;
	p.r = RADIUS;
	Vector2f ort = RandomOrt();
	p.ort[0] = ort(0);
	p.ort[1] = ort(1);
	p.type = mt;
	matters.push_back(p);
	prevMatters.push_back(p);
}

void Environment::AddMatter(MatterType mt, float x, float y, float ortx, float orty)
{
	Matter p;
	p.pos[0] = x;
	p.pos[1] = y;
	p.v = V;
	p.r = RADIUS;
	p.ort[0] = ortx;
	p.ort[1] = orty;
	p.type = mt;
	matters.push_back(p);
	prevMatters.push_back(p);
}

void Environment::RemoveMatters()
{
	matters.clear();
	prevMatters.clear();
}

Environment::~Environment()
{
	RemoveMatters();
}